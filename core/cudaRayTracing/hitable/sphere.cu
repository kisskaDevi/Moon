#include "hip/hip_runtime.h"
#include "sphere.h"

__device__ bool sphere::hit(const ray& r, float tMin, float tMax, hitRecord& rec) const {
    bool result = false;

    vec4 oc = r.getOrigin() - center;
    float a = dot(r.getDirection(), r.getDirection());
    float b = dot(oc, r.getDirection()) / a;
    float c = dot(oc, oc) - radius * radius / a;
    float discriminant = b * b - c;

    if (discriminant >= 0) {
        discriminant = sqrt(discriminant);
        float temp = -b - discriminant;
        result = (temp < tMax && temp > tMin);
        if (!result) {
            temp = -b + discriminant;
            result = (temp < tMax && temp > tMin);
        }
        if (result) {
            rec.t = temp;
            rec.point = r.point(rec.t);
            rec.normal = (rec.point - center) / radius;
            rec.color = color;
            rec.props = props;
        }
    }
    return result;
}

__global__ void createSphere(sphere** sph, vec4 cen, float r, vec4 color, const properties props) {
    *sph = new sphere(cen, r, color, props);
}

sphere* sphere::create(vec4 cen, float r, vec4 color, const properties& props) {
    sphere** sph;
    checkCudaErrors(hipMalloc((void**)&sph, sizeof(sphere**)));

    createSphere<<<1,1>>>(sph, cen, r, color, props);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    sphere* hostsph = new sphere;
    checkCudaErrors(hipMemcpy(&hostsph, sph, sizeof(sphere*), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(sph));

    return hostsph;
}
