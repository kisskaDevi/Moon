#include "hip/hip_runtime.h"
#include "cudaRayTracing.h"
#include "operations.h"
#include "ray.h"
#include "material.h"

namespace cuda::rayTracing {

RayTracing::RayTracing(){}
RayTracing::~RayTracing(){}


__global__ void initCurandState(size_t width, size_t height, hiprandState* randState)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (int pixel = j * width + i; (i < width) && (j < height)) {
        hiprand_init(clock64(), pixel, 0, &randState[pixel]);
    }
}

void RayTracing::create()
{
    record = Buffer<FrameRecord>(width * height);
    baseColor = Buffer<uint32_t>(width * height);
    bloomColor = Buffer<uint32_t>(width * height);
    randState = Buffer<hiprandState>(width * height);

    dim3 blocks(width / xThreads + 1, height / yThreads + 1, 1);
    dim3 threads(xThreads, yThreads, 1);
    initCurandState<<<blocks, threads>>>(width, height, randState.get());
}

void RayTracing::buildTree(){
    hostContainer.makeTree();

    devContainer = make_devicep<Container_dev>(Container_dev());
    add(devContainer.get(), extractHitables(hostContainer.storage));

    if(std::is_same<Container_dev, HitableKDTree>::value){
        const auto linearSizes = hostContainer.getLinearSizes();
        const auto linearBoxes = hostContainer.getLinearBoxes();
        Buffer<uint32_t> devNodeCounter(linearSizes.size(), (uint32_t*) linearSizes.data());
        Buffer<box> devNodeBox(linearBoxes.size(), (box*) linearBoxes.data());
        makeTree((HitableKDTree*)devContainer.get(), devNodeCounter.get(), devNodeBox.get(), linearSizes.size());
    }
}

__device__ bool isEmit(const HitRecord& rec){
    return (rec.rayDepth == 1 && rec.vertex.props.emissionFactor >= 0.98f) || (rec.scattering.getDirection().length2() > 0.0f && rec.lightIntensity >= 0.95f);
}

struct FrameBuffer {
    vec4f base{0.0f};
    vec4f bloom{0.0f};
};

template<typename ContainerType>
__device__ FrameBuffer getFrame(uint32_t minRayIterations, uint32_t maxRayIterations, Camera* cam, float u, float v, HitRecord& rec, ContainerType* container, hiprandState* randState) {
    FrameBuffer result;
    do {
        ray r = rec.rayDepth++ ? rec.scattering : cam->getPixelRay(u, v, randState);
        if (HitCoords coords; container->hit(r, coords)) {
            if(vec4 color = rec.vertex.color; coords.check()){
                coords.obj->calcHitRecord(r, coords, rec);
                rec.lightIntensity *= rec.vertex.props.absorptionFactor;
                rec.vertex.color = min(
                    vec4f(rec.lightIntensity * rec.vertex.color.x(),
                          rec.lightIntensity * rec.vertex.color.y(),
                          rec.lightIntensity * rec.vertex.color.z(),
                          rec.vertex.color.a()),
                    color);
            }
        }

        vec4f scattering = scatter(r, rec.vertex.normal, rec.vertex.props, randState);
        if(scattering.length2() == 0.0f || rec.rayDepth >= maxRayIterations){
            result.base = rec.vertex.props.emissionFactor >= 0.98f ? rec.vertex.props.emissionFactor * rec.vertex.color : vec4f(0.0f, 0.0f, 0.0f, 1.0f);
            result.bloom = isEmit(rec) ? rec.vertex.color : vec4f(0.0f, 0.0f, 0.0f, 0.0f);
            rec = HitRecord{};
            break;
        }
        rec.scattering = ray(rec.vertex.point, scattering);
    } while (rec.rayDepth < minRayIterations);
    return result;
}

template <typename ContainerType>
__global__ void render(bool clear, size_t width, size_t height, size_t minRayIterations, size_t maxRayIterations, uint32_t* baseColor, uint32_t* bloomColor, FrameRecord* record, Camera* cam, ContainerType* container, hiprandState* randState)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (int pixel = j * width + i; (i < width) && (j < height)) {
        // hiprandState randState;
        // hiprand_init(clock64(), pixel, 0, &randState);

        float u = 1.0f - 2.0f * float(i) / float(width);
        float v = 2.0f * float(j) / float(height) - 1.0f;

        if(clear){
            record[pixel] = FrameRecord{};
        }

        FrameBuffer frame = getFrame(minRayIterations, maxRayIterations, cam, u, v, record[pixel].hit, container, &randState[pixel]);
        record[pixel].color += frame.base;
        record[pixel].bloom += frame.bloom;

        vec4f base = record[pixel].color / ::max(1.0f, record[pixel].color.a());
        baseColor[pixel] = uint32_t(255.0f*base[2]) << 0 | uint32_t(255.0f*base[1]) << 8 | uint32_t(255.0f*base[0]) << 16 | uint32_t(255) << 24;
        vec4f bloom = record[pixel].bloom / ::max(1.0f, record[pixel].bloom.a());
        bloomColor[pixel] = uint32_t(255.0f*bloom[2]) << 0 | uint32_t(255.0f*bloom[1]) << 8 | uint32_t(255.0f*bloom[0]) << 16 | uint32_t(255) << 24;
    }
}

__global__ void updateKernel(Camera* cam){
    cam->update();
}

void RayTracing::update(){
    updateKernel<<<1, 1>>>(cam->get());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

bool RayTracing::calculateImage(uint32_t* hostBaseColor, uint32_t* hostBloomColor)
{
    dim3 blocks(width / xThreads + 1, height / yThreads + 1, 1);
    dim3 threads(xThreads, yThreads, 1);
    render<<<blocks, threads>>>(
        clear,
        width,
        height,
        minRayIterations,
        maxRayIterations,
        baseColor.get(),
        bloomColor.get(),
        record.get(),
        cam->get(),
        devContainer.get(),
        randState.get());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    clear = false;

    checkCudaErrors(hipMemcpy(hostBaseColor, baseColor.get(), sizeof(uint32_t) * baseColor.getSize(), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hostBloomColor, bloomColor.get(), sizeof(uint32_t) * bloomColor.getSize(), hipMemcpyDeviceToHost));

    return true;
}

}
