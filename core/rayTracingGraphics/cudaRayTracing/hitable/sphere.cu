#include "hip/hip_runtime.h"
#include "sphere.h"
#include "operations.h"

namespace cuda {

__host__ __device__ void sphere::calcBox() {
    bbox.min = center - vec4(radius, radius, radius, 0.0f);
    bbox.max = center + vec4(radius, radius, radius, 0.0f);
}

__host__ __device__ sphere::sphere(vec4 cen, float r, vec4 color, const properties& props) : center(cen), radius(r), color(color), props(props) {
    calcBox();
}

__host__ __device__ sphere::sphere(vec4 cen, float r, vec4 color) : center(cen), radius(r), color(color) {
    calcBox();
}

__host__ __device__ bool sphere::hit(const ray& r, float tMin, float tMax, hitCoords& coord) const {
    vec4 oc = r.getOrigin() - center;
    float a = 1.0f / r.getDirection().length2();
    float b = - dot(oc, r.getDirection()) * a;
    float c = oc.length2() - radius * radius * a;
    float discriminant = b * b - c;

    if (discriminant < 0) {
        return false;
    }

    discriminant = sqrt(discriminant);
    float temp = b - discriminant;
    bool result = (temp < tMax && temp > tMin);
    if (!result) {
        temp = b + discriminant;
        result = (temp < tMax && temp > tMin);
    }
    if (result) {
        coord = {temp, 0.0f, 0.0f};
    }
    return result;
}

__host__ __device__ void sphere::calcHitRecord(const ray& r, const hitCoords& coord, hitRecord& rec) const {
    rec.point = r.point(coord.t);
    rec.normal = (rec.point - center) / radius;
    rec.color = color;
    rec.props = props;
}

__global__ void createSphere(sphere** sph, vec4 cen, float r, vec4 color, const properties props) {
    *sph = new sphere(cen, r, color, props);
}

sphere* sphere::create(vec4 cen, float r, vec4 color, const properties& props) {
    sphere** sph;
    checkCudaErrors(hipMalloc((void**)&sph, sizeof(sphere**)));

    createSphere<<<1,1>>>(sph, cen, r, color, props);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    sphere* hostsph = nullptr;
    checkCudaErrors(hipMemcpy(&hostsph, sph, sizeof(sphere*), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(sph));

    return hostsph;
}

sphere* sphere::create(const sphere* pHost){
    return sphere::create(pHost->center, pHost->radius, pHost->color, pHost->props);
}

}
