#include "hip/hip_runtime.h"
#include "sphere.h"
#include "operations.h"

namespace cuda {

__host__ __device__ sphere::sphere(const vec4f& cen, float r, const vec4f& color, const properties& props) : center(cen), radius(r), color(color), props(props) {}

__host__ __device__ sphere::sphere(const vec4f& cen, float r, const vec4f& color) : center(cen), radius(r), color(color) {}

__host__ __device__ bool sphere::hit(const ray& r, hitCoords& coord) const {
    vec4f oc = r.getOrigin() - center;
    float a = 1.0f / r.getDirection().length2();
    float b = - dot(oc, r.getDirection()) * a;
    float c = oc.length2() - radius * radius * a;
    float discriminant = b * b - c;

    if (discriminant < 0) {
        return false;
    }

    discriminant = sqrt(discriminant);
    float temp = b - discriminant;
    bool result = (temp < coord.tmax && temp > coord.tmin);
    if (!result) {
        temp = b + discriminant;
        result = (temp < coord.tmax && temp > coord.tmin);
    }
    if (result) {
        coord.tmax = temp;
    }
    return result;
}

__host__ __device__ void sphere::calcHitRecord(const ray& r, const hitCoords& coord, hitRecord& rec) const {
    rec.point = r.point(coord.tmax);
    rec.normal = (rec.point - center) / radius;
    rec.color = color;
    rec.props = props;
}

__global__ void createKernel(sphere* sph, vec4f cen, float r, vec4f color, const properties props) {
    sph = new (sph) sphere(cen, r, color, props);
}

void sphere::create(sphere* dpointer, const sphere& host){
    createKernel<<<1,1>>>(dpointer, host.center, host.radius, host.color, host.props);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void destroyKernel(sphere* p) {
    p->~sphere();
}

void sphere::destroy(sphere* dpointer){
    destroyKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__host__ __device__ box sphere::calcBox() const {
    box bbox;
    bbox.min = center - vec4f(radius, radius, radius, 0.0f);
    bbox.max = center + vec4f(radius, radius, radius, 0.0f);
    return bbox;
}
}
