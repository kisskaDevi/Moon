#include "camera.h"

__host__ __device__ void cuda::camera::update(){
    horizontal = aspect * vec4::getHorizontal(viewRay.getDirection());
    vertical = vec4::getVertical(viewRay.getDirection());
}

__host__ __device__ cuda::camera::camera(){}

__host__ __device__ cuda::camera::camera(
    const ray viewRay,
    float aspect,
    float matrixScale,
    float matrixOffset,
    float focus,
    float apertura) :
    viewRay(viewRay),
    aspect(aspect),
    matrixScale(matrixScale),
    matrixOffset(matrixOffset),
    focus(focus),
    apertura(apertura)
{
    update();
}

__host__ __device__ cuda::camera::camera(const ray& viewRay, float aspect) : viewRay(viewRay), aspect(aspect){
    update();
}

__device__ ray cuda::camera::getPixelRay(float u, float v, hiprandState* local_rand_state) const {
    const float t = focus / (matrixOffset - focus);
    u = matrixScale * t * u + apertura * float(hiprand_uniform(local_rand_state));
    v = matrixScale * t * v + apertura * float(hiprand_uniform(local_rand_state));
    return ray(viewRay.point(matrixOffset), t * matrixOffset * viewRay.getDirection() - (u * horizontal + v * vertical));
}

__host__ __device__ ray cuda::camera::getPixelRay(float u, float v) const {
    const float t = focus / (matrixOffset - focus);
    u = matrixScale * t * u;
    v = matrixScale * t * v;
    return ray(viewRay.point(matrixOffset), t * matrixOffset * viewRay.getDirection() - (u * horizontal + v * vertical));
}
