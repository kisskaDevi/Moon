#include "hip/hip_runtime.h"
#include "camera.h"
#include "operations.h"

__host__ __device__ cuda::camera::camera(){}

__host__ __device__ cuda::camera::camera(
    const ray viewRay,
    float aspect,
    float matrixScale,
    float matrixOffset,
    float focus,
    float apertura) :
    viewRay(viewRay),
    aspect(aspect),
    matrixScale(matrixScale),
    matrixOffset(matrixOffset),
    focus(focus),
    apertura(apertura)
{
    horizontal = aspect * vec4::getHorizontal(viewRay.getDirection());
    vertical = vec4::getVertical(viewRay.getDirection());
}

__host__ __device__ cuda::camera::camera(const ray viewRay, float aspect) : aspect(aspect){
    setViewRay(viewRay);
}

__device__ ray cuda::camera::getPixelRay(float u, float v, hiprandState* local_rand_state) {
    const float t = focus / (matrixOffset - focus);
    u = matrixScale * t * u + apertura * float(hiprand_uniform(local_rand_state));
    v = matrixScale * t * v + apertura * float(hiprand_uniform(local_rand_state));
    return ray(viewRay.point(matrixOffset), t * matrixOffset * viewRay.getDirection() - (u * horizontal + v * vertical));
}

__host__ __device__ ray cuda::camera::getPixelRay(float u, float v) {
    const float t = focus / (matrixOffset - focus);
    u = matrixScale * t * u;
    v = matrixScale * t * v;
    return ray(viewRay.point(matrixOffset), t * matrixOffset * viewRay.getDirection() - (u * horizontal + v * vertical));
}

__host__ __device__ ray cuda::camera::getViewRay(){
    return viewRay;
}

__host__ __device__ void cuda::camera::setViewRay(const ray& viewRay){
    this->viewRay = viewRay;
    horizontal = aspect * vec4::getHorizontal(viewRay.getDirection());
    vertical = vec4::getVertical(viewRay.getDirection());
}

__host__ __device__ void cuda::camera::setFocus(const float& focus){
    this->focus = focus;
}

cuda::camera* cuda::camera::create(const ray& viewRay, float aspect) {
    cuda::camera* cam = nullptr;
    cuda::camera hostcam(viewRay, aspect);
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(cuda::camera)));
    checkCudaErrors(hipMemcpy(cam, &hostcam, sizeof(cuda::camera), hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    return cam;
}

void cuda::camera::reset(camera* cam, const ray& viewRay, float aspect) {
    cuda::camera hostcam(viewRay, aspect);
    checkCudaErrors(hipMemcpy(cam, &hostcam, sizeof(cuda::camera), hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

void cuda::camera::destroy(camera* cam) {
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void setViewRayKernel(cuda::camera* cam, const ray viewRay){
    cam->setViewRay(viewRay);
}

void cuda::camera::setViewRay(camera* cam, const ray& viewRay){
    setViewRayKernel<<<1,1>>>(cam, viewRay);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void setFocusKernel(cuda::camera* cam, const float focus){
    cam->setFocus(focus);
}

void cuda::camera::setFocus(camera* cam, const float& focus){
    setFocusKernel<<<1,1>>>(cam, focus);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

cuda::camera cuda::camera::copyToHost(camera* pDevice){
    cuda::camera hostcam;
    checkCudaErrors(hipMemcpy(&hostcam, pDevice, sizeof(cuda::camera), hipMemcpyDeviceToHost));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    return hostcam;
}
