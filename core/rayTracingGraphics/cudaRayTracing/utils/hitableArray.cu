#include "hip/hip_runtime.h"
#include "hitableArray.h"

#include "operations.h"

__host__ __device__ hitableArray::~hitableArray() {
    for(size_t i = 0; i < size; i++){
        delete array[i];
    }
    delete[] array;
    size = 0;
}

__device__ bool hitableArray::hit(const ray& r, float tMin, float tMax, hitRecord& rec) const {
    hitCoords coord = {tMax, 0.0f, 0.0f};
    hitable* resObj = nullptr;
    for (size_t i = 0; i < size; i++) {
        if (array[i]->hit(r, tMin, coord.t, coord)) {
            resObj = array[i];
        }
    }
    if(coord.t != tMax && resObj){
        rec = resObj->calcHitRecord(r, coord);
        return true;
    }
    return false;
}

__host__ __device__ void hitableArray::add(hitable* object) {
    hitable** newArray = new hitable*[size + 1];
    for(size_t i = 0; i < size; i++){
        newArray[i] = array[i];
    }
    newArray[size] = object;
    delete[] array;
    array = newArray;
    size++;
}

__global__ void createArray(hitableArray** arr) {
    *arr = new hitableArray();
}

hitableArray* hitableArray::create() {
    hitableArray** array;
    checkCudaErrors(hipMalloc((void**)&array, sizeof(hitableArray**)));

    createArray<<<1, 1>>>(array);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hitableArray* hostarr = nullptr;
    checkCudaErrors(hipMemcpy(&hostarr, array, sizeof(hitableArray*), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(array));

    return hostarr;
}
