#include "hip/hip_runtime.h"
#include "hitableContainer.h"

#include "operations.h"

namespace cuda::rayTracing {

__global__ void addKernel(HitableContainer* container, Hitable** object, size_t size) {
    container->add(object, size);
}

void add(HitableContainer* container, std::vector<Hitable*>& objects) {
    addKernel<<<1,1>>>(container, objects.data(), objects.size());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void destroyKernel(HitableContainer* p) {
    p->~HitableContainer();
}

void HitableContainer::destroy(HitableContainer* dpointer){
    destroyKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

}
