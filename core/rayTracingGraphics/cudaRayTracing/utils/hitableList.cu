#include "hip/hip_runtime.h"
#include "hitableList.h"
#include "operations.h"

__host__ __device__ void destroyObject(hitable* object) {
    if (object->next) {
        destroyObject(object->next);
    }
    delete object;
}

__host__ __device__ hitableList::~hitableList() {
    destroyObject(head);
}

__device__ bool hitableList::hit(const ray& r, float tMin, float tMax, hitRecord& rec) const {
    hitCoords coord = {tMax, 0.0f, 0.0f};
    hitable* resObj = nullptr;
    for (hitable* object = head; object; object = object->next) {
        if (object->hit(r, tMin, coord.t, coord)) {
            resObj = object;
        }
    }
    if(coord.t != tMax && resObj){
        resObj->calcHitRecord(r, coord, rec);
        return true;
    }
    return false;
}

__host__ __device__ void hitableList::add(hitable* object) {
    if (head) {
        tail->next = object;
    } else {
        head = object;
        head->next = object;
    }
    tail = object;
}

__global__ void createList(hitableList** list) {
    *list = new hitableList();
}

hitableList* hitableList::create() {
    hitableList** list;
    checkCudaErrors(hipMalloc((void**)&list, sizeof(hitableList**)));

    createList<<<1, 1>>>(list);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hitableList* hostlist = nullptr;
    checkCudaErrors(hipMemcpy(&hostlist, list, sizeof(hitableList*), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(list));

    return hostlist;
}
