#include "hip/hip_runtime.h"
#include "utils/kdTree.h"
#include "utils/operations.h"
#include "utils/buffer.h"

namespace cuda::rayTracing {

__global__ void createTreeKernel(HitableKDTree* tree, uint32_t* offsets, box* boxes, HitableKDTree::KDNodeType* nodes)
{
    tree->makeTree(offsets, boxes, nodes);
}

void makeTree(HitableKDTree* container, uint32_t* offsets, box* boxes, size_t size){
    Buffer<HitableKDTree::KDNodeType> nodes(size);
    createTreeKernel<<<1,1>>>(container, offsets, boxes, nodes.release());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void createKernel(HitableKDTree* p) {
    p = new (p) HitableKDTree();
}

void HitableKDTree::create(HitableKDTree* dpointer, const HitableKDTree& host){
    createKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void destroyKernel(HitableKDTree* p) {
    p->~HitableKDTree();
}

void HitableKDTree::destroy(HitableKDTree* dpointer){
    destroyKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

}
