#include "hip/hip_runtime.h"
#include "utils/kdTree.h"

namespace cuda {

__global__ void createTreeKernel(kdTree* tree)
{
    tree->makeTree();
}

void makeTree(kdTree* container){
    createTreeKernel<<<1,1>>>(container);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void createKernel(kdTree* p) {
    p = new (p) kdTree();
}

void kdTree::create(kdTree* dpointer, const kdTree& host){
    createKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void destroyKernel(kdTree* p) {
    p->~kdTree();
}

void kdTree::destroy(kdTree* dpointer){
    destroyKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}
}
