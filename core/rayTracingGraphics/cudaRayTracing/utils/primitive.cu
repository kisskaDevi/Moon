#include "hip/hip_runtime.h"
#include "primitive.h"

#include <algorithm>

namespace cuda::rayTracing {

void sortByBox(std::vector<const Primitive*>::iterator begin, std::vector<const Primitive*>::iterator end, const box& bbox){
    const vec4f limits = bbox.max - bbox.min;
    std::sort(begin, end, [i = limits.maxValueIndex(3)](const Primitive* a, const Primitive* b){
        return a->getBox().min[i] < b->getBox().min[i];
    });
}

std::vector<Hitable*> extractHitables(const std::vector<const Primitive*>& storage){
    std::vector<Hitable*> hitables;
    for(const auto& p : storage){
        hitables.push_back(p->hit());
    }
    return hitables;
}

}
